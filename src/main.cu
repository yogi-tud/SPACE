#include <bitset>
#include <cstdlib>
#include <cstdint>
#include <iostream>
#include <filesystem>
#include <stdio.h>
#include <thread>
#include <vector>
#include <string>

#define DISABLE_CUDA_TIME
#define DELIM ";"
#include "cuda_time.cuh"
#include "cuda_try.cuh"

#include "csv_loader.hpp"
#include "utils.cuh"
#include "data_generator.cuh"
#include "benchmarks.cuh"



int main(int argc, char** argv)
{
    int dataset_pick=0;

    size_t datasize_MIB = 2048;
    size_t MEBIBYTE = (1<<20);
    float sel = 0.025;
    int iterations = 3;
    bool report_failures = true;
    string dataset = "";

    if (argc > 1) {
        int dataset_pick = atoi(argv[1]);
        printf("setting dataset to %i\n", dataset_pick);

    }
    if(argc > 2)
    {
         sel = atof(argv[2]);
        cout<<"SELECTIVITY: "<<sel<<endl;
    }
    if(argc > 3)
    {
        datasize_MIB  = atoi(argv[3]);
        cout<<"datasize: "<<datasize_MIB<<endl;
    }




    const char* csv_path = "../res/Arade_1.csv";

    // load data
    std::vector<float> col;
    size_t ele_count = MEBIBYTE* datasize_MIB / sizeof (uint64_t);

   // printf("parsing %s\n", csv_path);
    //

    //take 2. gpu default remove for other BENCHE!!!

    col.resize(ele_count);
    //0 uniform
    //1 skew
    //2 csv arade
    uint8_t* pred = (uint8_t*) malloc(col.size()*sizeof(uint8_t)); //mask on host

    // gen predicate mask
    size_t one_count=0;

    switch(dataset_pick) {
        case 0:
            genRandomInts(ele_count, 45000);

            generate_mask_uniform(pred, 0, col.size(), sel);
            one_count=col.size()*sel;
            dataset="uniform";
        break;
            one_count=col.size()*sel;
        case 1: generate_mask_zipf(pred,one_count,0,col.size());
            genRandomInts(ele_count, 45000);
            dataset="zipf";
        break;
        case 2: pred = gen_predicate(col, +[](float f) { return f > 55; }, &one_count).data();
            dataset="arade";
            load_csv(csv_path, {3}, col);
        case 3: generate_mask_burst(pred,one_count,0,col.size(),1);
            genRandomInts(ele_count, 45000);
            dataset="burst";
        break;
        default:generate_mask_uniform(pred, 0, col.size(), sel);
            break;
    }

    CUDA_TRY(hipSetDevice(1));

    float* d_input = vector_to_gpu(col);
    float* d_output = alloc_gpu<float>(col.size() + 1);

   // generate_mask_uniform( pred,0,col.size(),0.01);
    //generate_mask_zipf(pred,col.size()/1000,0,col.size());
    //  cpu_buffer_print(pred,0,1000);
    // auto pred = gen_predicate(
    //    col, +[](float f) { return f > 2000; }, &one_count);

    uint8_t* d_mask;
    size_t size = col.size()*sizeof(uint8_t);
    CUDA_TRY(hipMalloc(&d_mask, size));
    CUDA_TRY(hipMemcpy(d_mask, &pred[0], size, hipMemcpyHostToDevice));

    //uint8_t* d_mask = vector_to_gpu(pred);


    printf("line count: %zu, one count: %zu, percentage: %f\n", col.size(), one_count, (double)one_count / col.size());

    // gen cpu side validation
    std::vector<float> validation;
    validation.resize(col.size());
    size_t out_length = generate_validation(&col[0], &pred[0], &validation[0], col.size());
    float* d_validation = vector_to_gpu(validation);

    // prepare candidates for benchmark
    intermediate_data id{col.size(), 1024, 8}; // setup shared intermediate data

    //pair: experiment name, blocksize, gridsize, time ms

    std::vector<std::pair<std::string, float>> benchs;
    //timings for single kernels of a benchmark
    std::vector<std::pair<std::string, float>> subtimings;

    //building config string for csv output
    std::string gridblock ="";

    //set up benchmarks for different cuda configs and algorithm on same data set
        for(size_t blocksize = 1024; blocksize <=1024 ; blocksize = blocksize * 2 ) {
        for (size_t gridsize = 8096; gridsize <= 8096; gridsize = gridsize * 2) {



            gridblock = ";"+std::to_string(blocksize)+";"+std::to_string(gridsize);


            benchs.emplace_back("bench1_base_variant"+ gridblock

                                    , bench1_base_variant(&id, d_input, d_mask, d_output, col.size(), 1024, blocksize, gridsize));


            benchs.emplace_back(
                "bench2_base_variant_skipping"+gridblock, bench2_base_variant_skipping(&id, d_input, d_mask, d_output, col.size(), 1024, blocksize, gridsize));

            benchs.emplace_back("bench3_3pass_streaming"+gridblock, bench3_3pass_streaming(&id, d_input, d_mask, d_output, col.size(), 1024, blocksize, gridsize));

            benchs.emplace_back(
                "bench4_optimized_read_non_skipping_cub_pss"+gridblock,
                bench4_optimized_read_non_skipping_cub_pss(&id, d_input, d_mask, d_output, col.size(), 1024, blocksize, gridsize));

            benchs.emplace_back(
                "bench5_3pass_optimized_read_skipping_partial_pss"+gridblock,
                bench5_3pass_optimized_read_skipping_partial_pss(&id, d_input, d_mask, d_output, col.size(), 1024, blocksize, gridsize));

            benchs.emplace_back(
                "bench6_3pass_optimized_read_skipping_two_phase_pss"+gridblock,
                bench6_3pass_optimized_read_skipping_two_phase_pss(&id, d_input, d_mask, d_output, col.size(), 1024, blocksize, gridsize));

            benchs.emplace_back(
                "bench7_3pass_optimized_read_skipping_cub_pss"+gridblock,
                bench7_3pass_optimized_read_skipping_cub_pss(&id, d_input, d_mask, d_output, col.size(), 1024, blocksize, gridsize));
        }
    }

    //cub has static block/thread config
    benchs.emplace_back("bench8_cub_flagged",  bench8_cub_flagged(&id, d_input, d_mask, d_output, col.size()));

    //print subtimings:



    // run benchmark
    std::vector<float> timings(benchs.size(), 0.0f);
    for (int it = 0; it < iterations; it++) {
        for (size_t i = 0; i < benchs.size(); i++) {
            timings[i] += benchs[i].second;
            size_t failure_count;
            if (!validate(&id, d_validation, d_output, out_length, report_failures, &failure_count)) {
                fprintf(stderr, "validation failure in bench %s, run %i: %zu failures\n", benchs[i].first.c_str(), it, failure_count);
                // exit(EXIT_FAILURE);
            }
        }
    }
    std::cout<<"Number of experiments: "<<benchs.size()<<  std::endl;

     string current_path (std::filesystem::current_path());

    string device = "_rtx8000";
    //string filename = "../data/"+dataset+device+".txt";
   // string filename = "/home/fett/edbt/EDBT_2022/data"+dataset+device+".txt";
    string filename = current_path+"/"+dataset+device+".txt";
    //string filename = dataset+device+".txt";
    std::cout << "Current path is " << current_path << '\n'; // (1)



    write_bench_file(filename,benchs, timings,iterations ,col.size() ,dataset ,(double)one_count / col.size() );

  //  for (int i = 0; i < benchs.size(); i++) {
    //    std::cout<<subtimings[i].first<<" "<<subtimings[i].second<<std::endl;
        //std::cout << "benchmark " << benchs[i].first << " time (ms): " << (double)timings[i] / iterations << std::endl;
       // write_benchmark(col.size(),dataset,(double)one_count / col.size(),myfile,(double)timings[i] / iterations,benchs[i].first);

    //}
    return 0;
}




